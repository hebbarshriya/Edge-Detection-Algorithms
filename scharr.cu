#include "hip/hip_runtime.h"
%%writefile scharr.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float
clamp(float val, float minVal, float maxVal)
{
    return fmaxf(minVal, fminf(maxVal, val));
}

__global__ void scharrEdgeDetection(unsigned char *input, unsigned char *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int Gx[3][3]  = { { 3, 0, -3 }, { 10, 0, -10 }, { 3, 0, -3 } };
    int Gy[3][3] { { 3, 10, 3 }, { 0, 0, 0 }, { -3, -10, -3 } };;

    float edgeX = 0.0;
    float edgeY = 0.0;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
    {
        for (int ky = -1; ky <= 1; ky++)
        {
            for (int kx = -1; kx <= 1; kx++)
            {
                float val = (float)input[(y + ky) * width + (x + kx)];
                edgeX += Gx[ky + 1][kx + 1] * val;
                edgeY += Gy[ky + 1][kx + 1] * val;
            }
        }
    }

    float edge = sqrt(edgeX * edgeX + edgeY * edgeY);
    edge = clamp(edge, 0, 255);
    output[y * width + x] = (unsigned char)edge;
}
int main()
{
    int width, height, channels;
    unsigned char *img = stbi_load("Image.jpg", &width, &height, &channels, 1); // Load image and convert to grayscale
    clock_t start, end;
    double cpu_time_used;

    if (img == NULL)
    {
        fprintf(stderr, "Error in loading the image\n");
        exit(1);
    }

    unsigned char *dst = (unsigned char *)malloc(width * height);
    if (dst == NULL)
    {
        fprintf(stderr, "Memory allocation failed for output image\n");
        stbi_image_free(img);
        exit(1);
    }

    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, width * height);
    hipMalloc(&d_output, width * height);

    hipMemcpy(d_input, img, width * height, hipMemcpyHostToDevice);

    dim3 blocks((width + 15) / 16, (height + 15) / 16);
    dim3 threadsPerBlock(16, 16);

    start = clock();

    scharrEdgeDetection<<<blocks, threadsPerBlock>>>(d_input, d_output, width, height);
    hipDeviceSynchronize(); // Wait for the GPU to finish

    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Scharr filter on GPU took %f seconds to execute \n", cpu_time_used);

    hipMemcpy(dst, d_output, width * height, hipMemcpyDeviceToHost);

    stbi_write_png("output_scarr.png", width, height, 1, dst, width);

    hipFree(d_input);
    hipFree(d_output);
    stbi_image_free(img);
    free(dst);

    return 0;
}